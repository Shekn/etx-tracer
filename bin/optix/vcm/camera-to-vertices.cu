#include "hip/hip_runtime.h"
#include <etx/rt/shared/optix.hxx>
#include <etx/rt/shared/vcm_shared.hxx>

using namespace etx;

static __constant__ VCMGlobal global;

RAYGEN(main) {
  uint3 idx = optixGetLaunchIndex();
  auto& state = global.input_state[idx.x];
  if (state.ray_action_set()) {
    return;
  }

  const auto& scene = global.scene;
  const auto& options = global.options;
  const auto& light_vertices = global.light_vertices;
  const auto& light_paths = global.light_paths;
  auto& iteration = *global.iteration;

  Raytracing rt;
  vcm_connect_to_light_path(scene, iteration, light_paths, light_vertices, options, rt, state);
}
