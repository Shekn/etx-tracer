#include "hip/hip_runtime.h"
#include <etx/rt/shared/optix.hxx>
#include <etx/rt/shared/vcm_shared.hxx>

using namespace etx;

static __constant__ VCMGlobal global;

RAYGEN(main) {
  uint3 idx = optixGetLaunchIndex();
  auto& state = global.input_state[idx.x];
  if (state.ray_action_set()) {
    return;
  }

  const auto& scene = global.scene;
  const auto& options = global.options;
  const auto& light_vertices = global.light_vertices;
  const auto& light_paths = global.light_paths;
  auto& iteration = *global.iteration;

  Raytracing rt;
  vcm_update_camera_vcm(state);
  vcm_handle_direct_hit(scene, options, state);
  vcm_gather_vertices(scene, iteration, light_vertices, global.spatial_grid, options, state);

  vcm_connect_to_light(scene, iteration, options, rt, state);
  vcm_connect_to_light_path(scene, iteration, light_paths, light_vertices, options, rt, state);
}
