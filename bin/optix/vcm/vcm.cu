#include "hip/hip_runtime.h"
#include <etx/rt/shared/vcm_shared.hxx>

#if (ETX_NVCC_COMPILER == 0)
extern uint3 blockIdx;
extern uint3 blockDim;
extern uint3 threadIdx;
#endif

using namespace etx;

ETX_GPU_CALLABLE void gen_light_rays(VCMGlobal* global) {
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t index = x + y * global->scene.camera.image_size.x;
  if (index >= global->light_final_image.count)
    return;

  global->input_state[index] = vcm_generate_emitter_state(index, global->scene, *global->iteration);
  global->light_iteration_image[index] = {};
  global->iteration->active_paths = global->scene.camera.image_size.x * global->scene.camera.image_size.y;
}

ETX_GPU_CALLABLE void merge_light_image(VCMGlobal* global) {
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t index = x + y * global->scene.camera.image_size.x;
  if (index >= global->light_final_image.count)
    return;

  auto dst = global->light_final_image;
  auto src = global->light_iteration_image;
  float t = global->iteration->iteration / float(global->iteration->iteration + 1);
  dst[index] = lerp(src[index], dst[index], t);
}

ETX_GPU_CALLABLE void gen_camera_rays(VCMGlobal* global) {
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t index = x + y * global->scene.camera.image_size.x;
  if (index >= global->light_paths.count)
    return;

  const auto& light_path = global->light_paths[index];
  global->input_state[index] = vcm_generate_camera_state({x, y}, global->scene, *global->iteration, light_path.spect);
  global->camera_iteration_image[index] = {};
  global->iteration->active_paths = global->scene.camera.image_size.x * global->scene.camera.image_size.y;
}

ETX_GPU_CALLABLE void merge_camera_image(VCMGlobal* global) {
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t index = x + y * global->scene.camera.image_size.x;
  if (index >= global->camera_final_image.count)
    return;

  auto dst = global->camera_final_image;
  auto src = global->camera_iteration_image;
  float t = global->iteration->iteration / float(global->iteration->iteration + 1);
  dst[index] = lerp(src[index], dst[index], t);
}

ETX_GPU_CALLABLE void vcm_continue_camera_path(VCMGlobal* global_ptr) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= global_ptr->launch_dim)
    return;

  auto& global = *global_ptr;
  auto& state = global.input_state[idx];
  auto& iteration = *global.iteration;

  const auto& scene = global.scene;
  const auto& options = global.options;

  // Last kernel
  if (state.ray_action_set() == false) {
    bool continue_ray = vcm_next_ray(scene, PathSource::Camera, options, state, iteration);
    state.continue_ray(continue_ray);
  }

  if (state.should_continue_ray()) {
    int i = atomicAdd(&global.iteration->active_paths, 1u);
    global.output_state[i] = state;
  } else {
    float3 result = state.merged * global.iteration->vm_normalization + (state.gathered / spectrum::sample_pdf()).to_xyz();
    uint32_t x = state.global_index % global.scene.camera.image_size.x;
    uint32_t y = state.global_index / global.scene.camera.image_size.x;
    uint32_t c = x + (global.scene.camera.image_size.y - 1 - y) * global.scene.camera.image_size.x;
    float4& current = global.camera_iteration_image[c];
    atomicAdd(&current.x, result.x);
    atomicAdd(&current.y, result.y);
    atomicAdd(&current.z, result.z);
  }
}
